#include "hip/hip_runtime.h"
#include <NvInfer.h>
#include <NvInferRuntime.h>
#include <NvOnnxParser.h>

#include <hip/hip_runtime.h>

#include <stdio.h>
#include <math.h>

#include <unistd.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <unistd.h>
#include <functional>
#include <memory>
#include <assert.h>
#include <algorithm>
#include <opencv2/opencv.hpp>
#include <dirent.h>
#include <thread>
#include <mutex>
#include <condition_variable>

#include "cuda-tools.cuh"
#include "trt_builder.cuh"
#include "simple-logger.cuh"
#include "mix-memory.cuh"
#include "img-process.cuh"
#include "timer.cuh"

using namespace std;


inline const char* severity_string(nvinfer1::ILogger::Severity t){
    switch(t){
        case nvinfer1::ILogger::Severity::kINTERNAL_ERROR: return "internal_error";
        case nvinfer1::ILogger::Severity::kERROR:   return "error";
        case nvinfer1::ILogger::Severity::kWARNING: return "warning";
        case nvinfer1::ILogger::Severity::kINFO:    return "info";
        case nvinfer1::ILogger::Severity::kVERBOSE: return "verbose";
        default: return "unknow";
    }
}

// coco数据集的labels，关于coco：https://cocodataset.org/#home
static const char* cocolabels[] = {
    "person", "bicycle", "car", "motorcycle", "airplane",
    "bus", "train", "truck", "boat", "traffic light", "fire hydrant",
    "stop sign", "parking meter", "bench", "bird", "cat", "dog", "horse",
    "sheep", "cow", "elephant", "bear", "zebra", "giraffe", "backpack",
    "umbrella", "handbag", "tie", "suitcase", "frisbee", "skis",
    "snowboard", "sports ball", "kite", "baseball bat", "baseball glove",
    "skateboard", "surfboard", "tennis racket", "bottle", "wine glass",
    "cup", "fork", "knife", "spoon", "bowl", "banana", "apple", "sandwich",
    "orange", "broccoli", "carrot", "hot dog", "pizza", "donut", "cake",
    "chair", "couch", "potted plant", "bed", "dining table", "toilet", "tv",
    "laptop", "mouse", "remote", "keyboard", "cell phone", "microwave",
    "oven", "toaster", "sink", "refrigerator", "book", "clock", "vase",
    "scissors", "teddy bear", "hair drier", "toothbrush"
};

// hsv转bgr
static std::tuple<uint8_t, uint8_t, uint8_t> hsv2bgr(float h, float s, float v){
    const int h_i = static_cast<int>(h * 6);
    const float f = h * 6 - h_i;
    const float p = v * (1 - s);
    const float q = v * (1 - f*s);
    const float t = v * (1 - (1 - f) * s);
    float r, g, b;
    switch (h_i) {
    case 0:r = v; g = t; b = p;break;
    case 1:r = q; g = v; b = p;break;
    case 2:r = p; g = v; b = t;break;
    case 3:r = p; g = q; b = v;break;
    case 4:r = t; g = p; b = v;break;
    case 5:r = v; g = p; b = q;break;
    default:r = 1; g = 1; b = 1;break;}
    return make_tuple(static_cast<uint8_t>(b * 255), static_cast<uint8_t>(g * 255), static_cast<uint8_t>(r * 255));
}

static std::tuple<uint8_t, uint8_t, uint8_t> random_color(int id){
    float h_plane = ((((unsigned int)id << 2) ^ 0x937151) % 100) / 100.0f;;
    float s_plane = ((((unsigned int)id << 3) ^ 0x315793) % 100) / 100.0f;
    return hsv2bgr(h_plane, s_plane, 1);
}

class TRTLogger : public nvinfer1::ILogger{
public:
    virtual void log(Severity severity, nvinfer1::AsciiChar const* msg) noexcept override{
        if(severity <= Severity::kWARNING){
            // 打印带颜色的字符，格式如下：
            // printf("\033[47;33m打印的文本\033[0m");
            // 其中 \033[ 是起始标记
            //      47    是背景颜色
            //      ;     分隔符
            //      33    文字颜色
            //      m     开始标记结束
            //      \033[0m 是终止标记
            // 其中背景颜色或者文字颜色可不写
            // 部分颜色代码 https://blog.csdn.net/ericbar/article/details/79652086
            if(severity == Severity::kWARNING){
                printf("\033[33m%s: %s\033[0m\n", severity_string(severity), msg);
            }
            else if(severity <= Severity::kERROR){
                printf("\033[31m%s: %s\033[0m\n", severity_string(severity), msg);
            }
            else{
                printf("%s: %s\n", severity_string(severity), msg);
            }
        }
    }
} logger;


// 通过智能指针管理nv返回的指针参数
// 内存自动释放，避免泄漏
template<typename _T>
shared_ptr<_T> make_nvshared(_T* ptr){
    return shared_ptr<_T>(ptr, [](_T* p){p->destroy();});
}

bool exists(const string& path){

#ifdef _WIN32
    return ::PathFileExistsA(path.c_str());
#else
    return access(path.c_str(), R_OK) == 0;
#endif
}


vector<unsigned char> load_file(const string& file){
    ifstream in(file, ios::in | ios::binary);
    if (!in.is_open())
        return {};

    in.seekg(0, ios::end);
    size_t length = in.tellg();

    std::vector<uint8_t> data;
    if (length > 0){
        in.seekg(0, ios::beg);
        data.resize(length);

        in.read((char*)&data[0], length);
    }
    in.close();
    return data;
}


bool build_model(const char *path)
{   
    if(exists("workspace/engine.trtmodel")){
        printf("Engine.trtmodel has exists.\n");
        return true;
    }

    //SimpleLogger::set_log_level(SimpleLogger::LogLevel::Verbose);
    TRT::compile(
        TRT::Mode::FP32,
        512,
        path,
        "workspace/engine.trtmodel",
        1 << 28
    );
    INFO("Done.");
    return true;
}

void copy_toGPU(float* input_data_device_start, vector<vector<cv::Mat>> &imgMats, float *d2is_start, 
                vector<string> filePaths, const int buffer_id, const int batch_size)
{
    int real_batch_size   = filePaths.size();
    int input_channel = 3;
    int input_height  = 640;
    int input_width   = 640;
    int input_numel   = batch_size * input_channel * input_height * input_width;
    IMGPRrocess::encode_kernel_invoker(filePaths, input_data_device_start+buffer_id*input_numel, 
                                       imgMats[buffer_id], 
                                       d2is_start+6*batch_size*buffer_id, real_batch_size, 
                                       input_width, input_height);

}

void inference(float* input_data_device, vector<unsigned char> &engine_data,
               vector<cv::Mat> &imgMats, float *d2is, 
               const int input_batch){

    TRTLogger logger;
    auto runtime   = make_nvshared( nvinfer1::createInferRuntime(logger) );
    auto engine = make_nvshared( runtime->deserializeCudaEngine(engine_data.data(), engine_data.size()) );
    if(engine == nullptr){
        printf("Deserialize cuda engine failed.\n");
        return;
    }

    if(engine->getNbBindings() != 2){
        printf("onnx导出有问题, 必须是1个输入和1个输出, 你有：%d个输出.\n", engine->getNbBindings() - 1);
        return;
    }

    int input_channel = 3;
    int input_height  = 640;
    int input_width   = 640;
    int input_numel   = input_batch * input_channel * input_height * input_width;

    
    auto execution_context = make_nvshared( engine->createExecutionContext() );
    hipStream_t stream = nullptr;
    checkRuntime(hipStreamCreate(&stream));
    // 设置输出 3x3输入，对应3x3输出
    auto output_dims = engine->getBindingDimensions(1);

    int output_numbox = output_dims.d[1];
    int output_numprob = output_dims.d[2];
    int num_classes = output_numprob - 5;
    int output_numel = input_batch * output_numbox * output_numprob;

    MixMemory output_data;
    //float* output_data_host   = output_data.cpu<float>(output_numel);
    float* output_data_device = output_data.gpu<float>(output_numel);
    
    // 明确当前推理时，使用的数据输入大小
    auto input_dims = engine->getBindingDimensions(0);
    input_dims.d[0] = input_batch;

    execution_context->setBindingDimensions(0, input_dims);
    float* bindings[] = {input_data_device, output_data_device};
    bool success      = execution_context->enqueueV2((void**)bindings, stream, nullptr);
    checkRuntime(hipStreamSynchronize(stream));
    checkRuntime(hipStreamDestroy(stream));
    
    // decode box：从不同尺度下的预测还原到原输入图上(包括:预测框，类被概率，置信度
    
    float confidence_threshold = 0.25;
    float nms_threshold = 0.5;
    #if DEBUG
        std::cout<<"output_numbox------------------>"<<output_numbox<<std::endl;
    #endif
    for (int kk = 0; kk < input_batch; kk++)
    {
        vector<vector<float>> bboxes = 
                IMGPRrocess::decode_kernel_invoker(output_data_device+kk*output_numbox*output_numprob, 
                                                   d2is+6*kk, output_numbox, output_numprob, num_classes, 
                                                   confidence_threshold);
        #if DEBUG    
            printf("decoded bboxes.size = %d\n", bboxes.size());
        #endif
        // nms非极大抑制
        std::sort(bboxes.begin(), bboxes.end(), [](vector<float>& a, vector<float>& b){return a[5] > b[5];});
        std::vector<bool> remove_flags(bboxes.size());
        std::vector<vector<float>> box_result;
        box_result.reserve(bboxes.size());

        auto iou = [](const vector<float>& a, const vector<float>& b){
            float cross_left   = std::max(a[0], b[0]);
            float cross_top    = std::max(a[1], b[1]);
            float cross_right  = std::min(a[2], b[2]);
            float cross_bottom = std::min(a[3], b[3]);

            float cross_area = std::max(0.0f, cross_right - cross_left) * std::max(0.0f, cross_bottom - cross_top);
            float union_area = std::max(0.0f, a[2] - a[0]) * std::max(0.0f, a[3] - a[1]) 
                            + std::max(0.0f, b[2] - b[0]) * std::max(0.0f, b[3] - b[1]) - cross_area;
            if(cross_area == 0 || union_area == 0) return 0.0f;
            return cross_area / union_area;
        };

        for(int i = 0; i < bboxes.size(); ++i){
            if(remove_flags[i]) continue;

            auto& ibox = bboxes[i];
            box_result.emplace_back(ibox);
            for(int j = i + 1; j < bboxes.size(); ++j){
                if(remove_flags[j]) continue;

                auto& jbox = bboxes[j];
                if(ibox[4] == jbox[4]){
                    // class matched
                    if(iou(ibox, jbox) >= nms_threshold)
                        remove_flags[j] = true;
                }
            }
        }
        #if DEBUG  
            printf("box_result.size = %d\n", box_result.size());
        #endif
        for(int i = 0; i < box_result.size(); ++i){
            auto& ibox = box_result[i];
            float left = ibox[0];
            float top = ibox[1];
            float right = ibox[2];
            float bottom = ibox[3];
            int class_label = ibox[4];
            float confidence = ibox[5];
            cv::Scalar color;
            tie(color[0], color[1], color[2]) = random_color(class_label);
            cv::rectangle(imgMats[kk], cv::Point(left, top), cv::Point(right, bottom), color, 3);

            auto name      = cocolabels[class_label];
            auto caption   = cv::format("%s %.2f", name, confidence);
            int text_width = cv::getTextSize(caption, 0, 1, 2, nullptr).width + 10;
            cv::rectangle(imgMats[kk], cv::Point(left-3, top-33), cv::Point(left + text_width, top), color, -1);
            cv::putText(imgMats[kk], caption, cv::Point(left, top-5), 0, 1, cv::Scalar::all(0), 2, 16);
        }
        string wt_path = "output/image-draw_"+to_string(input_batch)+"_"+to_string(kk)+".jpg";
        cv::imwrite(wt_path, imgMats[kk]);
    }
    
}


vector<string> get_imgFiles(string folderPath)
{
    std::vector<std::string> fileNames;

    DIR *dir;
    struct dirent *entry;

    if ((dir = opendir(folderPath.c_str())) != NULL) {
        while ((entry = readdir(dir)) != NULL) {
            // 忽略 "." 和 ".." 目录
            if (std::string(entry->d_name) != "." && std::string(entry->d_name) != "..") {
                std::string filePath = folderPath + "/" + entry->d_name;
                fileNames.push_back(filePath);
            }
        }
        closedir(dir);
    } else {
        INFOV("无法打开目录");
    }

    return fileNames;
}

// 缓冲区的最大容量
const int BUFFER_SIZE = 2;

// 缓冲区的头部和尾部指针
int head = 0;
int tail = 0;

// 互斥锁，用于保护对缓冲区的访问
std::mutex mtx;

// 条件变量，用于通知生产者和消费者
std::condition_variable cvar;


// 判断缓冲区是否为空的函数
bool is_buffer_empty() {
    return head == tail;
}

// 判断缓冲区是否为满的函数
bool is_buffer_full() {
    return (tail + 1) % BUFFER_SIZE == head;
}


// 生产者线程的函数
void producer(float* buffer, vector<vector<cv::Mat>> &imgMats, float *d2is_start, 
              vector<string> img_paths, vector<int> &buffer_batchSize, 
              const int batchSize, const int counts) 
{
    int count = 0;
    while (true) {
        std::unique_lock<std::mutex> lock(mtx);
        cvar.wait(lock, []{return !is_buffer_full();});
        vector<string> filePaths;
        cout<<"produce"<<endl;
        for (int i = count; i < count + batchSize && i < img_paths.size(); ++i) {
            filePaths.push_back(img_paths[i]);
        }
        int real_batchSize = filePaths.size();
        buffer_batchSize[tail] = real_batchSize;
        copy_toGPU(buffer, imgMats, d2is_start, filePaths, tail, batchSize);

        tail = (tail + 1) % BUFFER_SIZE;
        lock.unlock();
        cvar.notify_one();
        count += real_batchSize;
        if (count >= counts) {
            break;
        }
    }
}

// 消费者线程的函数
void consumer(float* input_data_device_start, vector<unsigned char> &engine_data,
              vector<vector<cv::Mat>> &imgMats, float *d2is_start, vector<int> &buffer_batchSize,
              const int batch_d2i_size, const int batch_img_size, const int counts) {
    int count = 0;
    while (true) {
        std::unique_lock<std::mutex> lock(mtx);
        cvar.wait(lock, []{return !is_buffer_empty();});
        cout<<"consum"<<endl;
        const int input_batch = buffer_batchSize[head];
        inference(input_data_device_start+head*batch_img_size, engine_data, imgMats[head], 
                  d2is_start+head*batch_d2i_size, input_batch);
        head = (head + 1) % BUFFER_SIZE;
        lock.unlock();
        cvar.notify_one();
        count += input_batch;
        if (count >= counts) {
            break;
        }
    }
}




int main(){
    const char *path = "/home/srb/trtLearning/yolo-dbintegrate/workspace/replaced.onnx";
    const char *engine_path = "/home/srb/trtLearning/yolo-dbintegrate/workspace/engine.trtmodel";
    string dir_Path = "/home/srb/trtLearning/images/VOC2005_1/PNGImages/TUGraz_cars";
    vector<string> img_paths = get_imgFiles(dir_Path);
    if (!access(engine_path, F_OK) == 0)
    {
        if(!build_model(path)){
            return -1;
        }
    }
    
    vector<unsigned char> engine_data = load_file(engine_path);

    int batchSize = 256; 
    int input_channel = 3;
    int input_height  = 640;
    int input_width   = 640;
    int input_numel   = batchSize * input_channel * input_height * input_width;
    const int batch_d2i_size = batchSize*6;
    vector<int> buffer_batchSize(BUFFER_SIZE);

    // 共享的数组作为缓冲区
    float* input_data_device_start;
    float *d2is_start = new float[BUFFER_SIZE*batchSize*6];
    checkRuntime( hipMalloc((void**)&input_data_device_start, BUFFER_SIZE*input_numel*sizeof(float)) );

    
    GpuTimer timer;
    float time_cost;
    timer.Start();

    vector<vector<cv::Mat>> imgMats(BUFFER_SIZE, vector<cv::Mat>(batchSize));
    const int counts = img_paths.size();
    std::thread t1(producer, input_data_device_start, std::ref(imgMats), d2is_start, 
                   img_paths, std::ref(buffer_batchSize),  batchSize, counts);
    std::thread t2(consumer, input_data_device_start, std::ref(engine_data),
                   std::ref(imgMats), d2is_start, std::ref(buffer_batchSize), batch_d2i_size, 
                   input_numel, counts);
    t1.join();
    t2.join();

    timer.Stop();
    time_cost = timer.Elapsed();
    printf("yolo-integrate: %f msecs.\n", time_cost);

    hipFree(input_data_device_start);
    delete[] d2is_start;
    return 0;
}
